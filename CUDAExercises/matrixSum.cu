#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 8
#define BLOCK_SIZE 2

__global__ void matrixSum(float *d_a, float *d_b, float *d_c){
    int globalIndex = blockIdx.y * BLOCK_SIZE * N + blockIdx.x * BLOCK_SIZE + threadIdx.y * N + threadIdx.x;

    d_c[globalIndex] = d_a[globalIndex] + d_b[globalIndex];
}

int main(){
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    int memSize = N * N * sizeof(float);

    h_a = (float*) malloc(memSize);
    h_b = (float*) malloc(memSize);
    h_c = (float*) malloc(memSize);

    hipMalloc((void**) &d_a, memSize);
    hipMalloc((void**) &d_b, memSize);
    hipMalloc((void**) &d_c, memSize);

    for (int i = 0; i < N * N; ++i) {
        h_a[i] = h_b[i] = 1.0f;
    }

    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, memSize, hipMemcpyHostToDevice);

    dim3 block(N / BLOCK_SIZE, N / BLOCK_SIZE);
    dim3 thread(BLOCK_SIZE, BLOCK_SIZE);

    matrixSum<<< block, thread >>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, memSize, hipMemcpyDeviceToHost);
    printf("El resultado es: \n");

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", h_c[i]);
        }
        printf("\n");
    }



    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}