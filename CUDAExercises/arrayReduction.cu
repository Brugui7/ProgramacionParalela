#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024


__global__ void arrayReduction(float *d_array, float *d_result){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) d_result[0] += d_array[tid];
}
int main() {
    float *h_array, *h_result;
    float *d_array, *d_result;
    int memSize = sizeof(float) * N;

    h_array = (float*) malloc(memSize);
    h_result = (float*) malloc(sizeof(float));

    hipError_t error;
    error = hipMalloc((void**)&d_array, memSize);
    if (error != hipSuccess){
        fprintf(stderr, "Error al reservar memoria");
        return -1;
    }

    error = hipMalloc((void**)&d_result, sizeof(float));
    if (error != hipSuccess){
        fprintf(stderr, "Error al reservar memoria");
        return -1;
    }

    //Fills the arrays
    for (int i = 0; i < N; ++i) {
        h_array[i] = 1.0f;
    }

    //Transfers
    error = hipMemcpy(d_array, h_array, memSize, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Error al transferir información.");
    }
    error = hipMemcpy(d_result, h_result, sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Error al transferir información.");
    }

    dim3 block (N/256);
    dim3 thread (256);

    arrayReduction<<<block, thread>>>(d_array, d_result);

    hipMemcpy(h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    printf("El resultado es: %f\n", h_result[0]);

    hipFree(d_array);

}