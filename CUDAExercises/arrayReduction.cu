/**
 * @author Alejandro Brugarolas
 * @since 2019-12
 */
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024


__global__ void arrayReduction(float *d_array){
    int idx = threadIdx.x;
    int idx2 = 0;

    for (int i = blockDim.x; i >= 1 ; i /=2) {
        if (idx < i){
            idx2 = idx + i;
            d_array[idx] += d_array[idx2];
        }
        __syncthreads();
    }
}
int main() {
    float *h_array;
    float *d_array;
    int memSize = sizeof(float) * N;

    h_array = (float*) malloc(memSize);

    hipError_t error;
    error = hipMalloc((void**)&d_array, memSize);
    if (error != hipSuccess){
        fprintf(stderr, "Error al reservar memoria");
        return -1;
    }


    //Fills the array
    for (int i = 0; i < N; ++i) {
        h_array[i] = 1.0f;
    }

    //Transfers
    error = hipMemcpy(d_array, h_array, memSize, hipMemcpyHostToDevice);
    if (error != hipSuccess){
        fprintf(stderr, "Error al transferir información.");
    }

    dim3 block (N / (N/2));
    dim3 thread (N/2);

    arrayReduction<<<block, thread>>>(d_array);

    hipMemcpy(h_array, d_array, sizeof(float), hipMemcpyDeviceToHost);

    printf("El resultado es: %f\n", h_array[0]);

    hipFree(d_array);

}