#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define N 1024

__global__ void saxpy(float *d_x, float *d_y){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N) d_y[tid] = d_x[tid] * 2.0f + d_y[tid];
}

int main(){
    float *h_y, *h_x;
    float *d_y, *d_x;
    int memSize = sizeof(float) * N;
    h_y = (float*) malloc(memSize);
    h_x = (float*) malloc(memSize);
    hipMalloc((void**)&d_x, memSize);
    hipMalloc((void**)&d_y, memSize);

    for (int i = 0; i < N; ++i) {
        h_x[i] = h_y[i] = 1.0f;
    }

    hipMemcpy(d_x, h_x, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, memSize, hipMemcpyHostToDevice);

    dim3 block(N / 256);
    dim3 thread(256);
    saxpy<<< block, thread >>>(d_x, d_y);

    hipMemcpy(h_x, d_x, memSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_y, d_y, memSize, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        printf("%f\n", h_y[i]);
    }

    free(h_y);
    free(h_x);
    hipFree(d_x);
    hipFree(d_y);


}